
#include <hip/hip_runtime.h>
#include <stdio.h>

/*
* kernel identified with __global__ function
* __global__ mean the function can be called by host
*/
__global__
void helloWorld()
{
    printf("Hello world from block:%d and thread:%d\n", blockIdx.x, threadIdx.x);
}

int main()
{
    size_t number_of_blocks, number_of_threads;

    number_of_blocks = 4;
    number_of_threads = 4;

    // Running kernel with configuration number_of_blocks and number_of_threads
    helloWorld<<<number_of_blocks, number_of_threads>>>();

    // Command before cudaDeviceSynchronize() will be executed when kernel is running
    printf("Hello world from Host\n");

    hipDeviceSynchronize();

    printf("Hello world from Host again\n");

    return 0;
}